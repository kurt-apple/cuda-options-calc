#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>

// Constants
#define M_PI            3.14159265f
#define TRADING_DAYS    252.0

// Calibration
#define BUCKET_SIZE         0.25f // TODO: implement dynamic bucket size based on strike width or other factors
#define BUCKET_LOW          5.0f // TODO: sizing based on standard deviation (derived from monte carlo simulation?) and/or personal conviction
#define BUCKET_HIGH         200.0f
#define SIMULATIONS_COUNT   10000 // TODO: it breaks 100k, why?

// Code Stuff
#define BUCKET_ARRAY_LENGTH (int) ((BUCKET_HIGH - BUCKET_LOW)/BUCKET_SIZE)                  // TODO: use a collection sizeable at run time
                                                                                            // TODO: store macro formulas with constant value in tmp vars
#define RANDSTATETYPE       hiprandState*                                                    // TODO: might not be useful anymore
#define SIMS_SIZE(x)        (SIMULATIONS_COUNT   * sizeof(x))                               // TODO: store macro formulas with constant value in tmp vars
#define BUCKETS_SIZE(x)     (BUCKET_ARRAY_LENGTH * sizeof(x))                               // TODO: store macro formulas with constant value in tmp vars
#define THREADS_PER_BLOCK   1024
#define NUM_BLOCKS          (SIMULATIONS_COUNT + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK // TODO: store macro formulas with constant value in tmp vars

// Option Stats
#define DTE          45.0  // TODO: support calendars and diagonals
#define STRIKE_PRICE 105.0 // TODO: support multiple option legs
#define PREMIUM      5.55
#define IS_PUT       true
#define IS_BUY       false

// Underlying Stats
#define UNDERLYING_PRICE 105.0
#define SIGMA            0.46

// Market/Economy Stats
#define RISK_FREE_RATE 0.05

// Formulas
#define YTE DTE/TRADING_DAYS

// Run GPU Code With Error Check
#define gpuErrorCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

__global__ void initRandomGenerator(RANDSTATETYPE state, unsigned long long seed)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(seed, tid, 0, &state[tid]);
}

__global__ void monteCarloSimulation(float* prices, float S, float T, float r, float sigma, RANDSTATETYPE state, int qty_simulations = SIMULATIONS_COUNT) // TODO: double or float
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    double price = S; // TODO: better variable naming
    double dt = T / TRADING_DAYS; // TODO: this doesn't look right

    // Each thread performs its own simulation and places result into prices array
    for (int i = tid; i < qty_simulations; i += blockDim.x * gridDim.x) { // TODO: what is required to run >10000 simulations? More VRAM?
        // Generate a random price path for the underlying asset
        price = S;

        for (double t = 0; t < T; t += dt) {
            double2 randomNums;
            randomNums.x = hiprand_uniform(&state[tid]); // TODO: switch to curand_uniform_double2 (didn't work before)
            randomNums.y = hiprand_uniform(&state[tid]);
            double z;
            if (randomNums.x > 0) {
                z = sqrt(-2.0 * log(randomNums.x)) * cos(2.0 * M_PI * randomNums.y); // TODO: Validate this formula
            }
            else z = 0;
            double drift = (r - 0.5 * sigma * sigma) * dt;
            double diffusion = sigma * sqrt(dt) * z;
            price *= exp(drift + diffusion);
        }

        // Store the final price in the array
        prices[i] = static_cast<float>(price); // Convert back to float
    }
}

double option_pl_at_expiry(double underlying_price, double strike_price, double premium, bool is_put, bool is_buy) {
    if (is_buy) { // TODO: fix formatting to look better
        if (is_put) return fmax(strike_price - underlying_price, 0.0) - premium; // TODO: don't trust this until I sit down and give a real answer
        return fmax(underlying_price - strike_price, 0.0) - premium; // TODO: don't trust this until I sit down and give a real answer
    }
    if (is_put) {
        if (underlying_price >= strike_price) return premium;
        return premium - (strike_price - underlying_price);
    }
    return fmin(premium, strike_price - underlying_price); // TODO: don't trust this until I sit down and give a real answer
}

void option_pl_array_populate(double strike_price, double premium, bool is_put, bool is_buy, double *pl_array, int len = BUCKET_ARRAY_LENGTH) {
    double tmp_price = BUCKET_LOW;
    for (int i = 0; i < len; i++) {
        pl_array[i] = option_pl_at_expiry(tmp_price, strike_price, premium, is_put, is_buy);
        tmp_price += BUCKET_SIZE;
    }
}

int main()
{
    // Allocate memory on the host for price results
    float* prices = (float*)malloc(SIMS_SIZE(float));

    double* optionPL = (double*)malloc(BUCKETS_SIZE(double));

    // Allocate memory on the device for price results
    float* devPrices;
    gpuErrorCheck(hipMalloc((void**)&devPrices, SIMS_SIZE(float)));

    // Allocate memory on the device for random number generation states
    RANDSTATETYPE devStates;
    gpuErrorCheck(hipMalloc((void**)&devStates, SIMS_SIZE(RANDSTATETYPE)));

    // Choose which GPU to run on, change this on a multi-GPU system.
    gpuErrorCheck(hipSetDevice(0));

    // Launch the kernel
    int numBlocks = (SIMULATIONS_COUNT + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    initRandomGenerator << <numBlocks, THREADS_PER_BLOCK >> > (devStates, time(NULL));
    monteCarloSimulation << <numBlocks, THREADS_PER_BLOCK >> > (devPrices, SIMULATIONS_COUNT, UNDERLYING_PRICE, yearsUntilExpiration, RISK_FREE_RATE, SIGMA, devStates);

    // Copy the results back to the host
    gpuErrorCheck(hipMemcpy(prices, devPrices, SIMS_SIZE(float), hipMemcpyDeviceToHost));

    //print generated prices
    /*for (int i = 0; i < SIMULATIONS_COUNT; i++) {
        printf("sim %i price: $%.2f", i, prices[i]);
    }*/

    float probabilities[BUCKET_ARRAY_SIZE];
    float current_price = BUCKET_LOW;
    float current_bucket_top;
    for (int i = 0; i < BUCKET_ARRAY_SIZE; i++) {
        current_bucket_top = current_price + BUCKET_SIZE;
        probabilities[i] = 0.0f;
        for (int j = 0; j < SIMULATIONS_COUNT; j++) {
            if (prices[j] >= current_price && prices[j] < current_bucket_top) {
                probabilities[i] += 1.0f;
            }
        }
        probabilities[i] /= SIMULATIONS_COUNT;
        current_price += BUCKET_SIZE;
    }

    current_price = BUCKET_LOW;

    option_pl_array_populate(STRIKEPRICE, PREMIUM, IS_PUT, IS_BUY, optionPL, BUCKET_ARRAY_SIZE);

    double total = 0.0;

    for (int i = 0; i < BUCKET_ARRAY_SIZE; i++) {
        printf("$%.2f to $%.2f expected return: $%.2f, probability: %.2f%%, factor: $%.2f\n", current_price, current_price + BUCKET_SIZE, optionPL[i], probabilities[i] * 100.0f, optionPL[i] * probabilities[i]);
        current_price += BUCKET_SIZE;
        total += optionPL[i] * probabilities[i];
    }

    std::cout << "total expected return: " << total << std::endl;

    // Free memory
    free(prices);
    free(optionPL);
    gpuErrorCheck(hipFree(devPrices));
    gpuErrorCheck(hipFree(devStates));

    std::cout << "Done." << std::endl;

    return 0;
}
